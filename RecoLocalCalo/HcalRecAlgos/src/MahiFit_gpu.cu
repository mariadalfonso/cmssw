#include "hip/hip_runtime.h"
#define EIGEN_NO_DEBUG  // kill throws in eigen code
#include "RecoLocalCalo/HcalRecAlgos/interface/MahiFit_gpu.h" 

namespace hcal { namespace mahi {

__device__
MahiFit::MahiFit(float const* pshape) :
  pshape_{pshape},
  functor_{pshape,
           false,false,false,
		   1,0,0,10}
{}

__device__
void MahiFit::phase1Apply(const HBHEChannelInfo& channelData,
			  float& reconstructedEnergy,
			  float& reconstructedTime,
			  bool& useTriple, 
			  float& chi2,
			  float* pulseNn, float* pulseMn, float* pulsePn,
			  float* pulseShapeArray, float* pulseDerivArray, float* pulseCovArray
			  ) const {

//  assert(channelData.nSamples()==8||channelData.nSamples()==10);

  resetWorkspace();

  nnlsWork_.tsOffset = channelData.soi();
  nnlsWork_.tsSize = channelData.nSamples();

  float reconstructedVals[3] = {0.f, -9999.f, -9999.f};
//  std::array<float,3> reconstructedVals {{ 0.0, -9999, -9999 }};
  
  double tsTOT = 0, tstrig = 0; // in GeV

  for(unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS){

    auto const amplitude = channelData.tsRawCharge(iTS) - channelData.tsPedestal(iTS);
    nnlsWork_.amplitudes.coeffRef(iTS) = amplitude;

    //ADC granularity
    auto const noiseADC = norm_ * channelData.tsDFcPerADC(iTS);

    //Electronic pedestal
    auto const pedWidth = channelData.tsPedestalWidth(iTS);

    //Photostatistics
    auto const noisePhoto = (amplitude > pedWidth) ? std::sqrt(amplitude * channelData.fcByPE()) : 0.f;

    //Total uncertainty from all sources
    nnlsWork_.noiseTerms.coeffRef(iTS) = noiseADC*noiseADC + noisePhoto*noisePhoto + pedWidth*pedWidth;

    tsTOT += amplitude;
    if (iTS == nnlsWork_.tsOffset)
        tstrig += amplitude;

    }

  tsTOT *= channelData.tsGain(0);
  tstrig *= channelData.tsGain(0);


  if(tstrig >= ts4Thresh_ && tsTOT > 0) {

    //Average pedestal width (for covariance matrix constraint)
    nnlsWork_.pedVal = 0.25f*( channelData.tsPedestalWidth(0)*channelData.tsPedestalWidth(0)+
			       channelData.tsPedestalWidth(1)*channelData.tsPedestalWidth(1)+
			       channelData.tsPedestalWidth(2)*channelData.tsPedestalWidth(2)+
			       channelData.tsPedestalWidth(3)*channelData.tsPedestalWidth(3) );

    // 1 sigma time constraint
    if (channelData.hasTimeInfo()) nnlsWork_.dt=timeSigmaSiPM_;
    else nnlsWork_.dt=timeSigmaHPD_;

    nnlsWork_.amplitudes.resize(nnlsWork_.tsSize);
    nnlsWork_.noiseTerms.resize(nnlsWork_.tsSize);

    useTriple=false;

    // only do pre-fit with 1 pulse if chiSq threshold is positive
    if (chiSqSwitch_>0) {
      doFit(reconstructedVals,1, pulseNn, pulseMn, pulsePn, pulseShapeArray, pulseDerivArray, pulseCovArray);
      if (reconstructedVals[2]>chiSqSwitch_) {
	doFit(reconstructedVals,0, pulseNn, pulseMn, pulsePn, pulseShapeArray, pulseDerivArray, pulseCovArray); //nbx=0 means use configured BXs
	useTriple=true;
      }
    }
    else {
      doFit(reconstructedVals,0, pulseNn, pulseMn, pulsePn, pulseShapeArray, pulseDerivArray, pulseCovArray);
      useTriple=true;
    }
  }
  else{
    reconstructedVals[0] = 0.; //energy
    reconstructedVals[1] = -9999.; //time
    reconstructedVals[2] = -9999.; //chi2
  }
  
  reconstructedEnergy = reconstructedVals[0]*channelData.tsGain(0);
  reconstructedTime = reconstructedVals[1];
  chi2 = reconstructedVals[2];

}

__device__
void MahiFit::doFit(float correctedOutput[3], int nbx, float* pulseNn, float* pulseMn, float* pulsePn, float* pulseShapeVector, float* pulseDerivVector, float* pulseCovVector) const {

  unsigned int bxSize=1;

  if (nbx==1) {
    nnlsWork_.bxOffset = 0;
  }
  else {
    bxSize = bxSizeConf_;
    nnlsWork_.bxOffset = static_cast<int>(nnlsWork_.tsOffset) >= bxOffsetConf_ ? bxOffsetConf_ : nnlsWork_.tsOffset;
  }

  nnlsWork_.nPulseTot = bxSize;

  if (dynamicPed_) nnlsWork_.nPulseTot++;
  nnlsWork_.bxs.setZero(nnlsWork_.nPulseTot);

  if (nbx==1) {
    nnlsWork_.bxs.coeffRef(0) = 0;
  }
  else {
    for (unsigned int iBX=0; iBX<bxSize; ++iBX) {
      nnlsWork_.bxs.coeffRef(iBX) = activeBXs_[iBX] - ((static_cast<int>(nnlsWork_.tsOffset) + activeBXs_[0]) >= 0 ? 0 : (nnlsWork_.tsOffset + activeBXs_[0]));
    }
  }

  nnlsWork_.maxoffset = nnlsWork_.bxs.coeff(bxSize-1);
  if (dynamicPed_) nnlsWork_.bxs[nnlsWork_.nPulseTot-1] = pedestalBX_;

  //  nnlsWork_.pulseMat.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);
  //  if(calculateArrivalTime_) nnlsWork_.pulseDerivMat.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);


  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int stride = blockDim.x*gridDim.x;
  if (idx >= stride ) return;

  SamplePulseMatrixMAP pulseMat_(pulseShapeVector + idx, DynStride(stride * MaxSVSize, stride));
  SamplePulseMatrixMAP pulseDerivMat_(pulseDerivVector + idx, DynStride(stride * MaxSVSize , stride));

  int sizeSQ=nnlsWork_.tsSize*nnlsWork_.tsSize;

  SampleMatrixMAP covs_[MaxPVSize] = {SampleMatrixMAP(pulseCovVector + idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + sizeSQ + idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 2*sizeSQ +idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 3*sizeSQ +idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 4*sizeSQ +idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 5*sizeSQ +idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 6*sizeSQ +idx, DynStride(stride * MaxSVSize, stride)),
				      SampleMatrixMAP(pulseCovVector + 7*sizeSQ +idx, DynStride(stride * MaxSVSize, stride))  };

  FullSampleVector pulseShapeArray;
  FullSampleVector pulseDerivArray;
  FullSampleMatrix pulseCov;

  int offset=0;
  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    offset=nnlsWork_.bxs.coeff(iBX);

    if (offset==pedestalBX_) {
      //      nnlsWork_.pulseMat.col(iBX) = SampleVector::Ones(nnlsWork_.tsSize);
      //      if(calculateArrivalTime_) nnlsWork_.pulseDerivMat.col(iBX) = SampleVector::Zero(nnlsWork_.tsSize);
    }
    else {

      pulseShapeArray.setZero();
      if(calculateArrivalTime_) pulseDerivArray.setZero();
      pulseCov.setZero();

      //      pulseShapeArray.setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);
      //      pulseDerivArray.setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);
      //      pulseCov.setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset,
      //      	     		nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);
      //      nnlsWork_.pulseCovArray[iBX].setZero(nnlsWork_.tsSize, nnlsWork_.tsSize);

      updatePulseShape(nnlsWork_.amplitudes.coeff(nnlsWork_.tsOffset + offset), 
		       pulseShapeArray,
		       pulseDerivArray,
		       pulseCov,
		       pulseNn, pulseMn, pulsePn
		       );
      

      //      nnlsWork_.pulseMat.col(iBX) = pulseShapeArray.segment(nnlsWork_.maxoffset - offset, nnlsWork_.tsSize);
      //      if(calculateArrivalTime_) nnlsWork_.pulseDerivMat.col(iBX) = pulseDerivArray.segment(nnlsWork_.maxoffset-offset, nnlsWork_.tsSize);
      //      nnlsWork_.pulseCovArray[iBX] = pulseCov.block(
      //         			    nnlsWork_.maxoffset - offset, nnlsWork_.maxoffset - offset, nnlsWork_.tsSize, nnlsWork_.tsSize);
      pulseMat_.col(iBX) = pulseShapeArray.segment(nnlsWork_.maxoffset - offset, nnlsWork_.tsSize);
      pulseDerivMat_.col(iBX) = pulseDerivArray.segment(nnlsWork_.maxoffset - offset, nnlsWork_.tsSize);
      covs_[iBX] = pulseCov.block(nnlsWork_.maxoffset - offset, nnlsWork_.maxoffset - offset, nnlsWork_.tsSize, nnlsWork_.tsSize);

    }
  }

  const float chiSq = minimize(pulseMat_,covs_);

  bool foundintime = false;
  unsigned int ipulseintime = 0;

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    if (nnlsWork_.bxs.coeff(iBX)==0) {
      ipulseintime = iBX;
      foundintime = true;
      break;
    }
  }

  if (foundintime) {
    correctedOutput[0] = nnlsWork_.ampVec.coeff(ipulseintime); //charge
    if (correctedOutput[0]!=0) {
        float arrivalTime = 0.f;
	if(calculateArrivalTime_) arrivalTime = calculateArrivalTime(pulseMat_,pulseDerivMat_,ipulseintime);
	correctedOutput[1] = arrivalTime; //time
    }
    else correctedOutput[1] = -9999.f;//time

    correctedOutput[2] = chiSq; //chi2

  }
}

__device__
const float MahiFit::minimize(SamplePulseMatrixMAP & pulseMat_, SampleMatrixMAP const * covs_) const {

  nnlsWork_.invcovp.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);
  nnlsWork_.ampVec.setZero(nnlsWork_.nPulseTot);
  //  nnlsWork_.aTaMat.setZero(nnlsWork_.nPulseTot, nnlsWork_.nPulseTot);
  //  nnlsWork_.aTbVec.setZero(nnlsWork_.nPulseTot);

  double oldChiSq=9999;
  double chiSq=oldChiSq;

  //  SampleMatrix invCovMat;
  //  invCovMat.setConstant(nnlsWork_.tsSize, nnlsWork_.tsSize, nnlsWork_.pedVal);
  //  invCovMat += nnlsWork_.noiseTerms.asDiagonal();

  for( int iter=1; iter<nMaxItersMin_ ; ++iter) {

    //    updateCov(invCovMat,covs_);
    updateCov(covs_);

    if (nnlsWork_.nPulseTot>1) {
      nnls(pulseMat_);
    }
    else {
      onePulseMinimize(pulseMat_);
    }

    const float newChiSq=calculateChiSq(pulseMat_);
    const float deltaChiSq = newChiSq - chiSq;

    if (newChiSq==oldChiSq && newChiSq<chiSq) {
      break;
    }
    oldChiSq=chiSq;
    chiSq = newChiSq;

    if (std::abs(deltaChiSq)<deltaChiSqThresh_) break;

  }

  return chiSq;

}

__device__
void MahiFit::updatePulseShape(const float itQ, FullSampleVector &pulseShape, FullSampleVector &pulseDeriv,
			       FullSampleMatrix &pulseCov,
			       float* pulseNn, float* pulseMn, float* pulsePn
			       ) const {
  
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int stride = blockDim.x*gridDim.x;

  float t0=meanTime_;

  if(applyTimeSlew_) {
    if(itQ<=1.0) t0+=tsDelay1GeV_;
    else t0+=0.f;
    // TODO: time slew has to be resolved eventually
    //hcalTimeSlewDelay_->delay(itQ,slewFlavor_);
  }


  float pulseN[HcalConst::maxSamples];
  float pulseM[HcalConst::maxSamples];
  float pulseP[HcalConst::maxSamples];

  for (unsigned int i=0; i<HcalConst::maxSamples; i++) {
    pulseNn[idx+stride*i] = 0.f;
    pulseMn[idx+stride*i] = 0.f;
    pulsePn[idx+stride*i] = 0.f;
  }

 const float xx = t0;
 const float xxm = -nnlsWork_.dt + t0;
 const float xxp = nnlsWork_.dt + t0;

//  (*pfunctor_)(&xx[0]);
  functor_.singlePulseShapeFunc(&xx);
  functor_.getPulseShape(pulseNn, idx, stride);
  //  functor_.getPulseShape(pulseN);

//  (*pfunctor_)(&xxm[0]);
  functor_.singlePulseShapeFunc(&xxm);
  functor_.getPulseShape(pulseMn, idx, stride);
  //  functor_.getPulseShape(pulseM);
  
//  (*pfunctor_)(&xxp[0]);
  functor_.singlePulseShapeFunc(&xxp);
  functor_.getPulseShape(pulsePn, idx, stride);
  //  functor_.getPulseShape(pulseP);

  //in the 2018+ case where the sample of interest (SOI) is in TS3, add an extra offset to align 
  //with previous SOI=TS4 case assumed by psfPtr_->getPulseShape()
  int delta = 4 - nnlsWork_.tsOffset;

  auto invDt = 0.5f / nnlsWork_.dt;

  for (unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS) {

    pulseShape[iTS+nnlsWork_.maxoffset] = pulseNn[idx+stride*(iTS+delta)];
    if(calculateArrivalTime_) pulseDeriv[iTS+nnlsWork_.maxoffset] = (pulseMn[idx+stride*(iTS+delta)]-pulsePn[idx+stride*(iTS+delta)])*invDt;

    pulseMn[idx+stride*(iTS+delta)] -= pulseNn[idx+stride*(iTS+delta)];
    pulsePn[idx+stride*(iTS+delta)] -= pulseNn[idx+stride*(iTS+delta)];

  }

  for (unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS) {
    for (unsigned int jTS=0; jTS<iTS+1; ++jTS) {

      auto const  tmp = 0.5 * ( pulsePn[idx+stride*(iTS+delta)]*pulsePn[idx+stride*(jTS+delta)] + pulseMn[idx+stride*(iTS+delta)]*pulseMn[idx+stride*(jTS+delta)] );
      pulseCov(iTS+nnlsWork_.maxoffset,jTS+nnlsWork_.maxoffset) = tmp;

    }
  }
  
}

__device__
//void MahiFit::updateCov(const SampleMatrix& samplecov, SampleMatrixMAP const * pulseCovArray_) const {
void MahiFit::updateCov(SampleMatrixMAP const * pulseCovArray_) const {

  //  SampleMatrix invCovMat=samplecov;
  SampleMatrix invCovMat;
  invCovMat.setConstant(nnlsWork_.tsSize, nnlsWork_.tsSize, nnlsWork_.pedVal);
  invCovMat += nnlsWork_.noiseTerms.asDiagonal();

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    auto const amp = nnlsWork_.ampVec.coeff(iBX);
    if (amp == 0) continue;

    int offset=nnlsWork_.bxs.coeff(iBX);

    if (offset==pedestalBX_) continue;		       
    else { 
      auto const ampsq = amp * amp;
      invCovMat += ampsq * pulseCovArray_[offset + nnlsWork_.bxOffset];
    }
  }

  nnlsWork_.covDecomp.compute(invCovMat);
}

__device__
float MahiFit::calculateArrivalTime(SamplePulseMatrixMAP & pulseMat_, SamplePulseMatrixMAP & pulseDerivMat_, unsigned int itIndex) const {

  if (nnlsWork_.nPulseTot > 1) {
    SamplePulseMatrix pulseDerivMatTMP = pulseDerivMat_;
    for (unsigned int iBX = 0; iBX < nnlsWork_.nPulseTot; ++iBX) {
      pulseDerivMat_.col(iBX) = pulseDerivMatTMP.col(nnlsWork_.bxs.coeff(iBX) + nnlsWork_.bxOffset);
    }
  }

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    pulseDerivMat_.col(iBX) *= nnlsWork_.ampVec.coeff(iBX);
  }

  SampleVector residuals = pulseMat_*nnlsWork_.ampVec - nnlsWork_.amplitudes;
  PulseVector solution = pulseDerivMat_.colPivHouseholderQr().solve(residuals);
  float t = solution.coeff(itIndex);
  t = (t>timeLimit_) ?  timeLimit_ : 
    ((t<-timeLimit_) ? -timeLimit_ : t);

  return t;

}
  
__device__
void MahiFit::nnls(SamplePulseMatrixMAP & pulseMat_) const {

  const unsigned int npulse = nnlsWork_.nPulseTot;
  const unsigned int nsamples = nnlsWork_.tsSize;

  nnlsWork_.invcovp = nnlsWork_.covDecomp.matrixL().solve(pulseMat_);
  nnlsWork_.aTaMat = nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.invcovp);
  nnlsWork_.aTbVec = nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.amplitudes));
  
  int iter = 0;
  Index idxwmax = 0;
  float wmax = 0.0;
  float threshold = nnlsThresh_;

  //  nnlsWork_.nP=0;
  
  while (true) {    
    if (iter>0 || nnlsWork_.nP==0) {
      if ( nnlsWork_.nP==std::min(npulse, nsamples)) break;
      
      const unsigned int nActive = npulse - nnlsWork_.nP;
      // exit if there are no more pulses to constrain
      if (nActive == 0)
        break;

      PulseVector updateWork = nnlsWork_.aTbVec - nnlsWork_.aTaMat*nnlsWork_.ampVec;
      
      Index idxwmaxprev = idxwmax;
      float wmaxprev = wmax;
      wmax = updateWork.tail(nActive).maxCoeff(&idxwmax);
      
      if (wmax<threshold || (idxwmax==idxwmaxprev && wmax==wmaxprev)) {
	break;
      }
      
      if (iter>=nMaxItersNNLS_) {
	break;
      }

      //unconstrain parameter
      Index idxp = nnlsWork_.nP + idxwmax;
      nnlsUnconstrainParameter(idxp, pulseMat_);

    }

    while (true) {
      if (nnlsWork_.nP==0) break;     

      PulseVector ampvecpermtest = nnlsWork_.ampVec.head(nnlsWork_.nP);
      
      solveSubmatrix(nnlsWork_.aTaMat,nnlsWork_.aTbVec,ampvecpermtest,nnlsWork_.nP);

      //check solution
      bool positive = true;
      for (unsigned int i = 0; i < nnlsWork_.nP; ++i)
        positive &= (ampvecpermtest(i) > 0);
      if (positive) {
        nnlsWork_.ampVec.head(nnlsWork_.nP) = ampvecpermtest.head(nnlsWork_.nP);
        break;
      } 

      //update parameter vector
      Index minratioidx=0;
      
      // no realizable optimization here (because it autovectorizes!)
      float minratio = std::numeric_limits<float>::max();
      for (unsigned int ipulse=0; ipulse<nnlsWork_.nP; ++ipulse) {
	if (ampvecpermtest.coeff(ipulse)<=0.) {
	  const float c_ampvec = nnlsWork_.ampVec.coeff(ipulse);
	  const float ratio = c_ampvec/(c_ampvec-ampvecpermtest.coeff(ipulse));
	  if (ratio<minratio) {
	    minratio = ratio;
	    minratioidx = ipulse;
	  }
	}
      }
      nnlsWork_.ampVec.head(nnlsWork_.nP) += minratio*(ampvecpermtest.head(nnlsWork_.nP) - nnlsWork_.ampVec.head(nnlsWork_.nP));
      
      //avoid numerical problems with later ==0. check
      nnlsWork_.ampVec.coeffRef(minratioidx) = 0.f;
      
      nnlsConstrainParameter(minratioidx, pulseMat_);
    }
   
    ++iter;

    //adaptive convergence threshold to avoid infinite loops but still
    //ensure best value is used
    if (iter%10==0) {
      threshold *= 10.;
    }
    
  }

  
}

__device__
void MahiFit::onePulseMinimize(SamplePulseMatrixMAP & pulseMat_) const {

  nnlsWork_.invcovp = nnlsWork_.covDecomp.matrixL().solve(pulseMat_);

  float aTaCoeff = (nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.invcovp)).coeff(0);

  float aTbCoeff = nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.amplitudes)).coeff(0);

  nnlsWork_.ampVec.coeffRef(0) = std::max(0.f, aTbCoeff / aTaCoeff);


}

__device__
float MahiFit::calculateChiSq(SamplePulseMatrixMAP & pulseMat_) const {
  
  return (nnlsWork_.covDecomp.matrixL().solve(pulseMat_*nnlsWork_.ampVec - nnlsWork_.amplitudes)).squaredNorm();
}

/*
__device__
void MahiFit::setPulseShapeTemplate(float const* pshape) {
//void MahiFit::setPulseShapeTemplate(const HcalPulseShapes::Shape& ps,const HcalTimeSlew* hcalTimeSlewDelay) {

  pshape_ = pshape;
  functor_.assign(pshape_,
                  false,false,false,
				  1,0,0,10);
  if (!(&ps == currentPulseShape_ ))
    {

      hcalTimeSlewDelay_ = hcalTimeSlewDelay;
      tsDelay1GeV_= hcalTimeSlewDelay->delay(1.0, slewFlavor_);

      resetPulseShapeTemplate(ps);
      currentPulseShape_ = &ps;
    }
}
*/

/*
__device__
void MahiFit::resetPulseShapeTemplate(const HcalPulseShapes::Shape& ps) { 
  ++ cntsetPulseShape_;

  // only the pulse shape itself from PulseShapeFunctor is used for Mahi
  // the uncertainty terms calculated inside PulseShapeFunctor are used for Method 2 only
  psfPtr_.reset(new FitterFuncs::PulseShapeFunctor(ps,false,false,false,
						   1,0,0,10));

}
*/

__device__
void MahiFit::nnlsUnconstrainParameter(Index idxp, SamplePulseMatrixMAP & pulseMat_) const {

  if (idxp != nnlsWork_.nP) {
    nnlsWork_.aTaMat.col(nnlsWork_.nP).swap(nnlsWork_.aTaMat.col(idxp));
    nnlsWork_.aTaMat.row(nnlsWork_.nP).swap(nnlsWork_.aTaMat.row(idxp));
    Eigen::numext::swap(nnlsWork_.aTbVec.coeffRef(nnlsWork_.nP),nnlsWork_.aTbVec.coeffRef(idxp));
    Eigen::numext::swap(nnlsWork_.ampVec.coeffRef(nnlsWork_.nP),nnlsWork_.ampVec.coeffRef(idxp)); // Victor did only this swap
    pulseMat_.col(nnlsWork_.nP).swap(pulseMat_.col(idxp));
    //  if(calculateArrivalTime_) pulseDerivMat_.col(nnlsWork_.nP).swap(pulseDerivMat_.col(idxp));
    Eigen::numext::swap(nnlsWork_.bxs.coeffRef(nnlsWork_.nP),nnlsWork_.bxs.coeffRef(idxp));
  }
  ++nnlsWork_.nP;

}

__device__
void MahiFit::nnlsConstrainParameter(Index minratioidx, SamplePulseMatrixMAP& pulseMat_) const {

  if (minratioidx != (nnlsWork_.nP - 1)) {
    nnlsWork_.aTaMat.col(nnlsWork_.nP-1).swap(nnlsWork_.aTaMat.col(minratioidx));
    nnlsWork_.aTaMat.row(nnlsWork_.nP-1).swap(nnlsWork_.aTaMat.row(minratioidx));
    Eigen::numext::swap(nnlsWork_.aTbVec.coeffRef(nnlsWork_.nP-1),nnlsWork_.aTbVec.coeffRef(minratioidx));
    Eigen::numext::swap(nnlsWork_.ampVec.coeffRef(nnlsWork_.nP-1),nnlsWork_.ampVec.coeffRef(minratioidx));
    pulseMat_.col(nnlsWork_.nP-1).swap(pulseMat_.col(minratioidx));
    //  if(calculateArrivalTime_) pulseDerivMat_.col(nnlsWork_.nP-1).swap(pulseDerivMat_.col(minratioidx));
    if (dynamicPed_ || calculateArrivalTime_) Eigen::numext::swap(nnlsWork_.bxs.coeffRef(nnlsWork_.nP-1),nnlsWork_.bxs.coeffRef(minratioidx));
  }
  --nnlsWork_.nP;

}

__device__
void MahiFit::solveSubmatrix(PulseMatrix& mat, PulseVector& invec, PulseVector& outvec, unsigned nP) const {
  using namespace Eigen;
  switch( nP ) { // pulse matrix is always square.
    /*
  case 10:
    {
      Matrix<double,10,10> temp = mat;
      outvec.head<10>() = temp.ldlt().solve(invec.head<10>());
    }
    break;
  case 9:
    {
      Matrix<double,9,9> temp = mat.topLeftCorner<9,9>();
      outvec.head<9>() = temp.ldlt().solve(invec.head<9>());
    }
    break;
    */
  case 8:
    {
      Matrix<float,8,8> temp = mat.topLeftCorner<8,8>();
      outvec.head<8>() = temp.ldlt().solve(invec.head<8>());
    }
    break;
  case 7:
    {
      Matrix<float,7,7> temp = mat.topLeftCorner<7,7>();
      outvec.head<7>() = temp.ldlt().solve(invec.head<7>());
    }
    break;
  case 6:
    {
      Matrix<float,6,6> temp = mat.topLeftCorner<6,6>();
      outvec.head<6>() = temp.ldlt().solve(invec.head<6>());
    }
    break;
  case 5:
    {
      Matrix<float,5,5> temp = mat.topLeftCorner<5,5>();
      outvec.head<5>() = temp.ldlt().solve(invec.head<5>());
    }
    break;
  case 4:
    {
      Matrix<float,4,4> temp = mat.topLeftCorner<4,4>();
      outvec.head<4>() = temp.ldlt().solve(invec.head<4>());
    }
    break;
  case 3: 
    {
      Matrix<float,3,3> temp = mat.topLeftCorner<3,3>();
      outvec.head<3>() = temp.ldlt().solve(invec.head<3>());
    }
    break;
  case 2:
    {
      Matrix<float,2,2> temp = mat.topLeftCorner<2,2>();
      outvec.head<2>() = temp.ldlt().solve(invec.head<2>());
    }
    break;
  case 1:
    {
      Matrix<float,1,1> temp = mat.topLeftCorner<1,1>();
      outvec.head<1>() = temp.ldlt().solve(invec.head<1>());
    }
    break;
  default:
    return;
  }
}

__device__
void MahiFit::resetWorkspace() const {

  nnlsWork_.nPulseTot=0;
  nnlsWork_.tsSize=0;
  nnlsWork_.tsOffset=0;
//  nnlsWork_.fullTSOffset=0;
  nnlsWork_.bxOffset=0;
  nnlsWork_.maxoffset=0;
  nnlsWork_.dt=0;
  nnlsWork_.nP=0;

  nnlsWork_.amplitudes.setZero();
  nnlsWork_.noiseTerms.setZero();
//  nnlsWork_.pedConstraint.setZero();


}


}}
