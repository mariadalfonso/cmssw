#include "hip/hip_runtime.h"
#include "RecoLocalCalo/HcalRecAlgos/interface/MahiFit_gpu.h" 

namespace hcal { namespace mahi {

__device__
MahiFit::MahiFit(float const* pshape) :
  fullTSSize_{19}, 
  fullTSofInterest_{8},
  pshape_{pshape},
  functor_{pshape,
           false,false,false,
		   1,0,0,10}
{}

__device__
void MahiFit::phase1Apply(const HBHEChannelInfo& channelData,
			  float& reconstructedEnergy,
			  float& reconstructedTime,
			  bool& useTriple, 
			  float& chi2) const {

//  assert(channelData.nSamples()==8||channelData.nSamples()==10);

  resetWorkspace();

  nnlsWork_.tsSize = channelData.nSamples();
  nnlsWork_.tsOffset = channelData.soi();
  nnlsWork_.fullTSOffset = fullTSofInterest_ - nnlsWork_.tsOffset;

  // 1 sigma time constraint
  if (channelData.hasTimeInfo()) nnlsWork_.dt=timeSigmaSiPM_;
  else nnlsWork_.dt=timeSigmaHPD_;


  //Average pedestal width (for covariance matrix constraint)
  float pedVal = 0.25*( channelData.tsPedestalWidth(0)*channelData.tsPedestalWidth(0)+
			channelData.tsPedestalWidth(1)*channelData.tsPedestalWidth(1)+
			channelData.tsPedestalWidth(2)*channelData.tsPedestalWidth(2)+
			channelData.tsPedestalWidth(3)*channelData.tsPedestalWidth(3) );

  nnlsWork_.pedConstraint.setConstant(nnlsWork_.tsSize, nnlsWork_.tsSize, pedVal);
  nnlsWork_.amplitudes.resize(nnlsWork_.tsSize);
  nnlsWork_.noiseTerms.resize(nnlsWork_.tsSize);

  float reconstructedVals[3] = {0.0, -9999, -9999};
//  std::array<float,3> reconstructedVals {{ 0.0, -9999, -9999 }};
  
  float tsTOT = 0, tstrig = 0; // in GeV
  for(unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS){
    float charge = channelData.tsRawCharge(iTS);
    float ped = channelData.tsPedestal(iTS);

    nnlsWork_.amplitudes.coeffRef(iTS) = charge - ped;

    //ADC granularity
    float noiseADC = (1./sqrt(12))*channelData.tsDFcPerADC(iTS);

    //Photostatistics
    float noisePhoto = 0;
    if ( (charge-ped)>channelData.tsPedestalWidth(iTS)) {
      noisePhoto = sqrt((charge-ped)*channelData.fcByPE());
    }

    //Electronic pedestal
    float pedWidth = channelData.tsPedestalWidth(iTS);

    //Total uncertainty from all sources
    nnlsWork_.noiseTerms.coeffRef(iTS) = noiseADC*noiseADC + noisePhoto*noisePhoto + pedWidth*pedWidth;

    tsTOT += (charge - ped)*channelData.tsGain(0);
    if( iTS==nnlsWork_.tsOffset ){
      tstrig += (charge - ped)*channelData.tsGain(0);
    }
  }

  if(tstrig >= ts4Thresh_ && tsTOT > 0) {

    useTriple=false;

    // only do pre-fit with 1 pulse if chiSq threshold is positive
    if (chiSqSwitch_>0) {
      doFit(reconstructedVals,1);
      if (reconstructedVals[2]>chiSqSwitch_) {
	doFit(reconstructedVals,0); //nbx=0 means use configured BXs
	useTriple=true;
      }
    }
    else {
      doFit(reconstructedVals,0);
      useTriple=true;
    }
  }
  else{
    reconstructedVals[0] = 0.; //energy
    reconstructedVals[1] = -9999.; //time
    reconstructedVals[2] = -9999.; //chi2
  }
  
  reconstructedEnergy = reconstructedVals[0]*channelData.tsGain(0);
  reconstructedTime = reconstructedVals[1];
  chi2 = reconstructedVals[2];

}

__device__
void MahiFit::doFit(float correctedOutput[3], int nbx) const {

  unsigned int bxSize=1;

  if (nbx==1) {
    nnlsWork_.bxOffset = 0;
  }
  else {
    bxSize = bxSizeConf_;
    nnlsWork_.bxOffset = static_cast<int>(nnlsWork_.tsOffset) >= bxOffsetConf_ ? bxOffsetConf_ : nnlsWork_.tsOffset;
  }

  nnlsWork_.nPulseTot = bxSize;

  if (dynamicPed_) nnlsWork_.nPulseTot++;
  nnlsWork_.bxs.setZero(nnlsWork_.nPulseTot);

  if (nbx==1) {
    nnlsWork_.bxs.coeffRef(0) = 0;
  }
  else {
    for (unsigned int iBX=0; iBX<bxSize; ++iBX) {
      nnlsWork_.bxs.coeffRef(iBX) = activeBXs_[iBX] - ((static_cast<int>(nnlsWork_.tsOffset) + activeBXs_[0]) >= 0 ? 0 : (nnlsWork_.tsOffset + activeBXs_[0]));
    }
  }

  nnlsWork_.maxoffset = nnlsWork_.bxs.coeffRef(bxSize-1);
  if (dynamicPed_) nnlsWork_.bxs[nnlsWork_.nPulseTot-1] = pedestalBX_;

  nnlsWork_.pulseMat.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);  
  nnlsWork_.pulseDerivMat.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);

  FullSampleVector pulseShapeArray;
  FullSampleVector pulseDerivArray;

  int offset=0;
  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    offset=nnlsWork_.bxs.coeff(iBX);

    pulseShapeArray.setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);
    pulseDerivArray.setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);
    nnlsWork_.pulseCovArray[iBX].setZero(nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset, nnlsWork_.tsSize + nnlsWork_.maxoffset + nnlsWork_.bxOffset);


    if (offset==pedestalBX_) {
      nnlsWork_.pulseMat.col(iBX) = SampleVector::Ones(nnlsWork_.tsSize);
    }
    else {

      updatePulseShape(nnlsWork_.amplitudes.coeff(nnlsWork_.tsOffset + offset), 
		       pulseShapeArray,
		       pulseDerivArray,
		       nnlsWork_.pulseCovArray[iBX]);
      

      nnlsWork_.pulseMat.col(iBX) = pulseShapeArray.segment(nnlsWork_.maxoffset - offset, nnlsWork_.tsSize);
      nnlsWork_.pulseDerivMat.col(iBX) = pulseDerivArray.segment(nnlsWork_.maxoffset-offset, nnlsWork_.tsSize);
    }
  }

  float chiSq = minimize();

  bool foundintime = false;
  unsigned int ipulseintime = 0;

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    if (nnlsWork_.bxs.coeff(iBX)==0) {
      ipulseintime = iBX;
      foundintime = true;
    }
  }

  if (foundintime) {
    correctedOutput[0] = nnlsWork_.ampVec.coeff(ipulseintime); //charge
    if (correctedOutput[0]!=0) {
	float arrivalTime = calculateArrivalTime();
	correctedOutput[1] = arrivalTime; //time
    }
    else correctedOutput[1] = -9999;//time

    correctedOutput[2] = chiSq; //chi2

  }
  
}

__device__
float MahiFit::minimize() const {

  nnlsWork_.invcovp.setZero(nnlsWork_.tsSize,nnlsWork_.nPulseTot);
  nnlsWork_.ampVec.setZero(nnlsWork_.nPulseTot);
  nnlsWork_.aTaMat.setZero(nnlsWork_.nPulseTot, nnlsWork_.nPulseTot);
  nnlsWork_.aTbVec.setZero(nnlsWork_.nPulseTot);

  float oldChiSq=9999;
  float chiSq=oldChiSq;

  for( int iter=1; iter<nMaxItersMin_ ; ++iter) {

    updateCov();

    if (nnlsWork_.nPulseTot>1) {
      nnls();
    }
    else {
      onePulseMinimize();
    }

    float newChiSq=calculateChiSq();
    float deltaChiSq = newChiSq - chiSq;

    if (newChiSq==oldChiSq && newChiSq<chiSq) {
      break;
    }
    oldChiSq=chiSq;
    chiSq = newChiSq;

    if (std::abs(deltaChiSq)<deltaChiSqThresh_) break;

  }

  return chiSq;

}

__device__
void MahiFit::updatePulseShape(float itQ, FullSampleVector &pulseShape, FullSampleVector &pulseDeriv,
			       FullSampleMatrix &pulseCov) const {
  
  float t0=meanTime_;

  if(applyTimeSlew_) {
    if(itQ<=1.0) t0+=tsDelay1GeV_;
    else t0+=0;
    // TODO: time slew has to be resolved eventually
    //hcalTimeSlewDelay_->delay(itQ,slewFlavor_);
  }


  double pulseN[MaxSVSize];
  double pulseM[MaxSVSize];
  double pulseP[MaxSVSize];

  for (unsigned int i=0; i<MaxSVSize; i++) {
      pulseN[i] = 0;
      pulseM[i] = 0;
      pulseP[i] = 0;
  }

  const double xx[4]={t0, 1.0, 0.0, 3};
  const double xxm[4]={-nnlsWork_.dt+t0, 1.0, 0.0, 3};
  const double xxp[4]={ nnlsWork_.dt+t0, 1.0, 0.0, 3};

//  (*pfunctor_)(&xx[0]);
  functor_.singlePulseShapeFunc(&xx[0]);
  functor_.getPulseShape(pulseN);

//  (*pfunctor_)(&xxm[0]);
  functor_.singlePulseShapeFunc(&xxm[0]);
  functor_.getPulseShape(pulseM);
  
//  (*pfunctor_)(&xxp[0]);
  functor_.singlePulseShapeFunc(&xxp[0]);
  functor_.getPulseShape(pulseP);

  //in the 2018+ case where the sample of interest (SOI) is in TS3, add an extra offset to align 
  //with previous SOI=TS4 case assumed by psfPtr_->getPulseShape()
  int delta = 4 - nnlsWork_.tsOffset;

  auto invDt = 0.25 / nnlsWork_.dt;

  for (unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS) {

    pulseShape.coeffRef(iTS+nnlsWork_.maxoffset) = pulseN[iTS+delta];
    pulseDeriv.coeffRef(iTS+nnlsWork_.maxoffset) = (pulseM[iTS+delta]-pulseP[iTS+delta])*invDt;

    pulseM[iTS] -= pulseN[iTS];
    pulseP[iTS] -= pulseN[iTS];
  }

  for (unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS) {
    for (unsigned int jTS=0; jTS<iTS+1; ++jTS) {
      
      float tmp = 0.5*( pulseP[iTS+delta]*pulseP[jTS+delta] +
			 pulseM[iTS+delta]*pulseM[jTS+delta] );

      pulseCov(iTS+nnlsWork_.maxoffset,jTS+nnlsWork_.maxoffset) += tmp;
      if(jTS!=iTS) pulseCov(jTS+nnlsWork_.maxoffset,iTS+nnlsWork_.maxoffset) += tmp;
    }
  }
  
}

__device__
void MahiFit::updateCov() const {

  SampleMatrix invCovMat;
  invCovMat.setZero(nnlsWork_.tsSize, nnlsWork_.tsSize);
  invCovMat = nnlsWork_.noiseTerms.asDiagonal();
  invCovMat +=nnlsWork_.pedConstraint;

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    if (nnlsWork_.ampVec.coeff(iBX)==0) continue;
    
    int offset=nnlsWork_.bxs.coeff(iBX);

    if (offset==pedestalBX_) continue;		       
    else { 
      invCovMat += nnlsWork_.ampVec.coeff(iBX)*nnlsWork_.ampVec.coeff(iBX)
	*nnlsWork_.pulseCovArray[offset+nnlsWork_.bxOffset].block(nnlsWork_.maxoffset-offset, nnlsWork_.maxoffset-offset, nnlsWork_.tsSize, nnlsWork_.tsSize);
    }
  }
  
  nnlsWork_.covDecomp.compute(invCovMat);
}

__device__
float MahiFit::calculateArrivalTime() const {

  int itIndex=0;

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    int offset=nnlsWork_.bxs.coeff(iBX);
    if (offset==0) itIndex=iBX;
  }

  PulseVector residuals = nnlsWork_.pulseMat*nnlsWork_.ampVec - nnlsWork_.amplitudes;
  PulseVector solution = nnlsWork_.pulseDerivMat.colPivHouseholderQr().solve(residuals);
  float t = solution.coeff(itIndex)/nnlsWork_.ampVec.coeff(itIndex);
  t = (t>timeLimit_) ?  timeLimit_ : 
    ((t<-timeLimit_) ? -timeLimit_ : t);

  return t;

}
  
__device__
void MahiFit::nnls() const {

  const unsigned int npulse = nnlsWork_.nPulseTot;
  const unsigned int nsamples = nnlsWork_.tsSize;

  PulseVector updateWork;
  updateWork.setZero(npulse);

  PulseVector ampvecpermtest;
  ampvecpermtest.setZero(npulse);

  nnlsWork_.invcovp = nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.pulseMat);
  nnlsWork_.aTaMat = nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.invcovp);
  nnlsWork_.aTbVec = nnlsWork_.invcovp.transpose().lazyProduct(nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.amplitudes));
  
  int iter = 0;
  Index idxwmax = 0;
  float wmax = 0.0;
  float threshold = nnlsThresh_;

  nnlsWork_.nP=0;
  
  while (true) {    
    if (iter>0 || nnlsWork_.nP==0) {
      if ( nnlsWork_.nP==std::min(npulse, nsamples)) break;
      
      const unsigned int nActive = npulse - nnlsWork_.nP;
      updateWork = nnlsWork_.aTbVec - nnlsWork_.aTaMat*nnlsWork_.ampVec;
      
      Index idxwmaxprev = idxwmax;
      float wmaxprev = wmax;
      wmax = updateWork.tail(nActive).maxCoeff(&idxwmax);
      
      if (wmax<threshold || (idxwmax==idxwmaxprev && wmax==wmaxprev)) {
	break;
      }
      
      if (iter>=nMaxItersNNLS_) {
	break;
      }

      //unconstrain parameter
      Index idxp = nnlsWork_.nP + idxwmax;
      nnlsUnconstrainParameter(idxp);

    }

    while (true) {
      if (nnlsWork_.nP==0) break;     

      ampvecpermtest = nnlsWork_.ampVec;
      
      solveSubmatrix(nnlsWork_.aTaMat,nnlsWork_.aTbVec,ampvecpermtest,nnlsWork_.nP);

      //check solution
      bool positive = true;
      for (unsigned int i = 0; i < nnlsWork_.nP; ++i)
        positive &= (ampvecpermtest(i) > 0);
      if (positive) {
        nnlsWork_.ampVec.head(nnlsWork_.nP) = ampvecpermtest.head(nnlsWork_.nP);
        break;
      } 

      //update parameter vector
      Index minratioidx=0;
      
      // no realizable optimization here (because it autovectorizes!)
      float minratio = std::numeric_limits<float>::max();
      for (unsigned int ipulse=0; ipulse<nnlsWork_.nP; ++ipulse) {
	if (ampvecpermtest.coeff(ipulse)<=0.) {
	  const float c_ampvec = nnlsWork_.ampVec.coeff(ipulse);
	  const float ratio = c_ampvec/(c_ampvec-ampvecpermtest.coeff(ipulse));
	  if (ratio<minratio) {
	    minratio = ratio;
	    minratioidx = ipulse;
	  }
	}
      }
      nnlsWork_.ampVec.head(nnlsWork_.nP) += minratio*(ampvecpermtest.head(nnlsWork_.nP) - nnlsWork_.ampVec.head(nnlsWork_.nP));
      
      //avoid numerical problems with later ==0. check
      nnlsWork_.ampVec.coeffRef(minratioidx) = 0.;
      
      nnlsConstrainParameter(minratioidx);
    }
   
    ++iter;

    //adaptive convergence threshold to avoid infinite loops but still
    //ensure best value is used
    if (iter%10==0) {
      threshold *= 10.;
    }
    
  }

  
}

__device__
void MahiFit::onePulseMinimize() const {

  nnlsWork_.invcovp = nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.pulseMat);

  SingleMatrix aTamatval = nnlsWork_.invcovp.transpose()*nnlsWork_.invcovp;
  SingleVector aTbvecval = nnlsWork_.invcovp.transpose()*nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.amplitudes);

  nnlsWork_.ampVec.coeffRef(0) = std::max(0.f, aTbvecval.coeff(0)/aTamatval.coeff(0));


}

__device__
float MahiFit::calculateChiSq() const {
  
  return (nnlsWork_.covDecomp.matrixL().solve(nnlsWork_.pulseMat*nnlsWork_.ampVec - nnlsWork_.amplitudes)).squaredNorm();
}

/*
__device__
void MahiFit::setPulseShapeTemplate(float const* pshape) {
//void MahiFit::setPulseShapeTemplate(const HcalPulseShapes::Shape& ps,const HcalTimeSlew* hcalTimeSlewDelay) {

  pshape_ = pshape;
  functor_.assign(pshape_,
                  false,false,false,
				  1,0,0,10);
  if (!(&ps == currentPulseShape_ ))
    {

      hcalTimeSlewDelay_ = hcalTimeSlewDelay;
      tsDelay1GeV_= hcalTimeSlewDelay->delay(1.0, slewFlavor_);

      resetPulseShapeTemplate(ps);
      currentPulseShape_ = &ps;
    }
}
*/

/*
__device__
void MahiFit::resetPulseShapeTemplate(const HcalPulseShapes::Shape& ps) { 
  ++ cntsetPulseShape_;

  // only the pulse shape itself from PulseShapeFunctor is used for Mahi
  // the uncertainty terms calculated inside PulseShapeFunctor are used for Method 2 only
  psfPtr_.reset(new FitterFuncs::PulseShapeFunctor(ps,false,false,false,
						   1,0,0,10));

}
*/

__device__
void MahiFit::nnlsUnconstrainParameter(Index idxp) const {
  nnlsWork_.aTaMat.col(nnlsWork_.nP).swap(nnlsWork_.aTaMat.col(idxp));
  nnlsWork_.aTaMat.row(nnlsWork_.nP).swap(nnlsWork_.aTaMat.row(idxp));
  nnlsWork_.pulseMat.col(nnlsWork_.nP).swap(nnlsWork_.pulseMat.col(idxp));
  nnlsWork_.pulseDerivMat.col(nnlsWork_.nP).swap(nnlsWork_.pulseDerivMat.col(idxp));
  Eigen::numext::swap(nnlsWork_.aTbVec.coeffRef(nnlsWork_.nP),nnlsWork_.aTbVec.coeffRef(idxp));
  Eigen::numext::swap(nnlsWork_.ampVec.coeffRef(nnlsWork_.nP),nnlsWork_.ampVec.coeffRef(idxp));
  Eigen::numext::swap(nnlsWork_.bxs.coeffRef(nnlsWork_.nP),nnlsWork_.bxs.coeffRef(idxp));
  ++nnlsWork_.nP;
}

__device__
void MahiFit::nnlsConstrainParameter(Index minratioidx) const {
  nnlsWork_.aTaMat.col(nnlsWork_.nP-1).swap(nnlsWork_.aTaMat.col(minratioidx));
  nnlsWork_.aTaMat.row(nnlsWork_.nP-1).swap(nnlsWork_.aTaMat.row(minratioidx));
  nnlsWork_.pulseMat.col(nnlsWork_.nP-1).swap(nnlsWork_.pulseMat.col(minratioidx));
  nnlsWork_.pulseDerivMat.col(nnlsWork_.nP-1).swap(nnlsWork_.pulseDerivMat.col(minratioidx));
  Eigen::numext::swap(nnlsWork_.aTbVec.coeffRef(nnlsWork_.nP-1),nnlsWork_.aTbVec.coeffRef(minratioidx));
  Eigen::numext::swap(nnlsWork_.ampVec.coeffRef(nnlsWork_.nP-1),nnlsWork_.ampVec.coeffRef(minratioidx));
  Eigen::numext::swap(nnlsWork_.bxs.coeffRef(nnlsWork_.nP-1),nnlsWork_.bxs.coeffRef(minratioidx));
  --nnlsWork_.nP;

}

__device__
void MahiFit::phase1Debug(const HBHEChannelInfo& channelData,
			  MahiDebugInfo& mdi) const {

  float recoEnergy, recoTime, chi2;
  bool use3;
  phase1Apply(channelData, recoEnergy, recoTime, use3, chi2);


  mdi.nSamples    = channelData.nSamples();
  mdi.soi         = channelData.soi();

  mdi.use3        = use3;

  mdi.inTimeConst = nnlsWork_.dt;
  mdi.inPedAvg    = 0.25*( channelData.tsPedestalWidth(0)*channelData.tsPedestalWidth(0)+
			   channelData.tsPedestalWidth(1)*channelData.tsPedestalWidth(1)+
			   channelData.tsPedestalWidth(2)*channelData.tsPedestalWidth(2)+
			   channelData.tsPedestalWidth(3)*channelData.tsPedestalWidth(3) );
  mdi.inGain      = channelData.tsGain(0);

  for (unsigned int iTS=0; iTS<channelData.nSamples(); ++iTS) {

    double charge = channelData.tsRawCharge(iTS);
    double ped = channelData.tsPedestal(iTS);

    mdi.inNoiseADC[iTS]  = (1./sqrt(12))*channelData.tsDFcPerADC(iTS);

    if ( (charge-ped)>channelData.tsPedestalWidth(iTS)) {
      mdi.inNoisePhoto[iTS] = sqrt((charge-ped)*channelData.fcByPE());
    }
    else { mdi.inNoisePhoto[iTS] = 0; }

    mdi.inPedestal[iTS]  = channelData.tsPedestalWidth(iTS);    
    mdi.totalUCNoise[iTS] = nnlsWork_.noiseTerms.coeffRef(iTS);

    if (channelData.hasTimeInfo()) {
      mdi.inputTDC[iTS] = channelData.tsRiseTime(iTS);
    }
    else { mdi.inputTDC[iTS]=-1; }

  }

  mdi.arrivalTime = recoTime;
  mdi.chiSq       = chi2;

  for (unsigned int iBX=0; iBX<nnlsWork_.nPulseTot; ++iBX) {
    if (nnlsWork_.bxs.coeff(iBX)==0) {
      mdi.mahiEnergy=nnlsWork_.ampVec.coeff(iBX);
      for(unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS){
	mdi.count[iTS] = iTS;
	mdi.inputTS[iTS] = nnlsWork_.amplitudes.coeff(iTS);
	mdi.itPulse[iTS] = nnlsWork_.pulseMat.col(iBX).coeff(iTS);
      }
    }
    else if (nnlsWork_.bxs.coeff(iBX)==pedestalBX_) {
      mdi.pedEnergy=nnlsWork_.ampVec.coeff(iBX);
    }
    else if (nnlsWork_.bxs.coeff(iBX)==-1) {
      mdi.pEnergy=nnlsWork_.ampVec.coeff(iBX);
      for(unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS){
        mdi.pPulse[iTS] = nnlsWork_.pulseMat.col(iBX).coeff(iTS);
      }
    }
    else if (nnlsWork_.bxs.coeff(iBX)==1) {
      mdi.nEnergy=nnlsWork_.ampVec.coeff(iBX);
      for(unsigned int iTS=0; iTS<nnlsWork_.tsSize; ++iTS){
	mdi.nPulse[iTS] = nnlsWork_.pulseMat.col(iBX).coeff(iTS);
      }
    }
  }  
}


__device__
void MahiFit::solveSubmatrix(PulseMatrix& mat, PulseVector& invec, PulseVector& outvec, unsigned nP) const {
  using namespace Eigen;
  switch( nP ) { // pulse matrix is always square.
  case 10:
    {
      Matrix<float,10,10> temp = mat;
      outvec.head<10>() = temp.ldlt().solve(invec.head<10>());
    }
    break;
  case 9:
    {
      Matrix<float,9,9> temp = mat.topLeftCorner<9,9>();
      outvec.head<9>() = temp.ldlt().solve(invec.head<9>());
    }
    break;
  case 8:
    {
      Matrix<float,8,8> temp = mat.topLeftCorner<8,8>();
      outvec.head<8>() = temp.ldlt().solve(invec.head<8>());
    }
    break;
  case 7:
    {
      Matrix<float,7,7> temp = mat.topLeftCorner<7,7>();
      outvec.head<7>() = temp.ldlt().solve(invec.head<7>());
    }
    break;
  case 6:
    {
      Matrix<float,6,6> temp = mat.topLeftCorner<6,6>();
      outvec.head<6>() = temp.ldlt().solve(invec.head<6>());
    }
    break;
  case 5:
    {
      Matrix<float,5,5> temp = mat.topLeftCorner<5,5>();
      outvec.head<5>() = temp.ldlt().solve(invec.head<5>());
    }
    break;
  case 4:
    {
      Matrix<float,4,4> temp = mat.topLeftCorner<4,4>();
      outvec.head<4>() = temp.ldlt().solve(invec.head<4>());
    }
    break;
  case 3: 
    {
      Matrix<float,3,3> temp = mat.topLeftCorner<3,3>();
      outvec.head<3>() = temp.ldlt().solve(invec.head<3>());
    }
    break;
  case 2:
    {
      Matrix<float,2,2> temp = mat.topLeftCorner<2,2>();
      outvec.head<2>() = temp.ldlt().solve(invec.head<2>());
    }
    break;
  case 1:
    {
      Matrix<float,1,1> temp = mat.topLeftCorner<1,1>();
      outvec.head<1>() = temp.ldlt().solve(invec.head<1>());
    }
    break;
  default:
    return;
  }
}

__device__
void MahiFit::resetWorkspace() const {

  nnlsWork_.nPulseTot=0;
  nnlsWork_.tsSize=0;
  nnlsWork_.tsOffset=0;
  nnlsWork_.fullTSOffset=0;
  nnlsWork_.bxOffset=0;
  nnlsWork_.maxoffset=0;
  nnlsWork_.dt=0;

  nnlsWork_.amplitudes.setZero();
  nnlsWork_.noiseTerms.setZero();
  nnlsWork_.pedConstraint.setZero();


}


}}
